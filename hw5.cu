

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cstdio>
#include <cstring>
#include <cassert>
#include <chrono>
#include <algorithm>
#include <vector>



// const int n_steps = 200000;
// const double dt = 60;
// const double eps = 1e-3;
// const double G = 6.674e-11;

#define n_steps 200000
#define dt 60
#define eps 1e-3
#define G 6.674e-11
#define planet_radius 1e7
#define missile_speed 1e6


__device__ 
double gravity_device_mass(double m0, double t) {
    return m0 + 0.5 * m0 * fabs(sin(t / 6000));
}

__device__
double get_missile_cost(double t) {
    return 1e5 + 1e3 * t; 
}



using namespace std::chrono;
using namespace std;

#define N_THRD_PER_BLK_X 4
#define N_THRD_PER_BLK_Y 32
#define N_THRD_PER_BLK (N_THRD_PER_BLK_X * N_THRD_PER_BLK_Y)


#define BODY_SIZE_BYTE 64 
#define BODY_SIZE_WORD 16 
#define BATCH_SIZE (N_THRD_PER_BLK_Y)
#define BATCH_SIZE_WORD (BATCH_SIZE * BODY_SIZE_WORD)

// need to make sure that this is int.
#define N_BODY_COPY_PER_PASS (N_THRD_PER_BLK * 4 / BODY_SIZE_BYTE) // (32 * 4 / 64) == 2.


typedef unsigned int WORD;
typedef unsigned char BYTE;

struct Body{
    
    double qx, qy, qz, vx, vy, vz, m;
    long long isDevice;
    
};

struct Input{
    int n;
    int planetId;
    int asteroidId;
    Body *bodyArray;
    int *id_map;
};





void read_input(const char* filename, Input *input) {

    std::ifstream fin(filename);
    fin >> input->n >> input->planetId >> input->asteroidId;

    input->bodyArray = new Body[input->n];
    input->id_map = new int[input->n];

    string type;

    for (int i = 0; i < input->n; i++) {
        fin >> input->bodyArray[i].qx 
            >> input->bodyArray[i].qy
            >> input->bodyArray[i].qz 
            >> input->bodyArray[i].vx 
            >> input->bodyArray[i].vy 
            >> input->bodyArray[i].vz 
            >> input->bodyArray[i].m 
            >> type;
        
        if (type != "device"){
            input->bodyArray[i].isDevice = 0;
        }
        else{
            input->bodyArray[i].isDevice = 1;
        }

        input->id_map[i] = i;
    }
}



__global__ void kernel_problem1(int step, int n_batch, int n, int planetId, int asteroidId,
                                Body *bodyArray, Body *bodyArray_update, BYTE *min_dist_sq){


    int bodyId_this = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    double ax = 0, ay = 0, az = 0, dx, dy, dz;
    double qx, qy, qz;

    
    // clock_t start_time = clock(); 
    // clock_t stop_time = clock();
    // int runtime = (int)(stop_time - start_time);

   
    if(bodyId_this < n){
        qx = bodyArray[bodyId_this].qx;
        qy = bodyArray[bodyId_this].qy;
        qz = bodyArray[bodyId_this].qz;
    }

    // update min_dist.
    if((bodyId_this == planetId) && (threadIdx.y == 0)){

        dx = qx - bodyArray[asteroidId].qx;
        dy = qy - bodyArray[asteroidId].qy;
        dz = qz - bodyArray[asteroidId].qz;

        *((double *)min_dist_sq) = min(*((double *)min_dist_sq), 
                                             dx * dx + dy * dy + dz * dz);  
    }

    __shared__ WORD sm[BATCH_SIZE_WORD + N_THRD_PER_BLK_Y * 3 * 2 * N_THRD_PER_BLK_X];
    double *sm_aggregate = (double *)(sm + BATCH_SIZE_WORD);


    for(int batchId = 0; batchId < n_batch; batchId++){

        
        for(int i = 0; i < BATCH_SIZE; i += N_BODY_COPY_PER_PASS){

            int global_offset = batchId * BATCH_SIZE_WORD;
            int local_offset = i * BODY_SIZE_WORD + tid;
            int idx = global_offset + local_offset;

            if(idx < n * BODY_SIZE_WORD){
                sm[local_offset] = ((WORD *)bodyArray)[idx];
            }
        }

        __syncthreads();

        int bodyId_other = batchId * BATCH_SIZE + threadIdx.y;
        
        if ((bodyId_other != bodyId_this) && (bodyId_other < n)){
            
            double mj = ((Body *)sm)[threadIdx.y].m;
     
            if (((Body *)sm)[threadIdx.y].isDevice == 1) {
                mj = gravity_device_mass(mj, step * dt);
            }

            dx = ((Body *)sm)[threadIdx.y].qx - qx;
            dy = ((Body *)sm)[threadIdx.y].qy - qy;
            dz = ((Body *)sm)[threadIdx.y].qz - qz;

            // if(bodyId_this == 0) start_time = clock();

            double dist3 = pow(dx * dx + dy * dy + dz * dz + eps * eps, 1.5);

            ax += G * mj * dx / dist3;    
            ay += G * mj * dy / dist3;    
            az += G * mj * dz / dist3; 

            // if(bodyId_this == 0){
            //     stop_time = clock();
            //     runtime = (int)(stop_time - start_time);
            //     printf("dt: %d\n", runtime);
            // } 
        }
    }

    sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 0] = ax * dt;
    sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 1] = ay * dt;
    sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 2] = az * dt;

    __syncthreads();

   
                      
    for(int binSize = 2; binSize <= blockDim.y; binSize = binSize << 1){

        if((threadIdx.y & (binSize - 1)) == 0){

            sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 0] += \
                sm_aggregate[(threadIdx.y + (binSize >> 1)) * (3 * blockDim.x) \
                        + 3 * threadIdx.x + 0];
            
            sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 1] += \
                sm_aggregate[(threadIdx.y + (binSize >> 1)) * (3 * blockDim.x) \
                        + 3 * threadIdx.x + 1];
            
            sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 2] += \
                sm_aggregate[(threadIdx.y + (binSize >> 1)) * (3 * blockDim.x) \
                        + 3 * threadIdx.x + 2];

        }
        __syncthreads();
    }



    if((threadIdx.y < 3) && (bodyId_this < n)){
        
        double *v_ptr = (double *)&(bodyArray[bodyId_this].vx);
        double *q_ptr_update = (double *)&(bodyArray_update[bodyId_this].qx);
        double *v_ptr_update = (double *)&(bodyArray_update[bodyId_this].vx);
        double *q_ptr_update_sm = sm_aggregate + (3 * blockDim.x + 3 * threadIdx.x);
   
        q_ptr_update_sm[0] = qx;
        q_ptr_update_sm[1] = qy;
        q_ptr_update_sm[2] = qz;
        
        double vi = v_ptr[threadIdx.y];
        vi += sm_aggregate[3 * threadIdx.x + threadIdx.y];

        v_ptr_update[threadIdx.y] = vi;
        q_ptr_update[threadIdx.y] = q_ptr_update_sm[threadIdx.y] + vi * dt;
        
    }
}







__global__ void kernel_problem2(int step, int n_batch, int n, int planetId, int asteroidId,
                                Body *bodyArray, Body *bodyArray_update, BYTE *hit_time_step){


    int bodyId_this = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    double ax = 0, ay = 0, az = 0, dx, dy, dz;
    double qx, qy, qz;

    
    // clock_t start_time = clock(); 
    // clock_t stop_time = clock();
    // int runtime = (int)(stop_time - start_time);

   
    if(bodyId_this < n){
        qx = bodyArray[bodyId_this].qx;
        qy = bodyArray[bodyId_this].qy;
        qz = bodyArray[bodyId_this].qz;
    }


    if((bodyId_this == planetId) && (threadIdx.y == 0)){

        dx = qx - bodyArray[asteroidId].qx;
        dy = qy - bodyArray[asteroidId].qy;
        dz = qz - bodyArray[asteroidId].qz;
        if (dx * dx + dy * dy + dz * dz < planet_radius * planet_radius) {
            if(*((int *)hit_time_step) == -2){
                *((int *)hit_time_step) = step - 1;
            }
        }
    }

    __shared__ WORD sm[BATCH_SIZE_WORD + N_THRD_PER_BLK_Y * 3 * 2 * N_THRD_PER_BLK_X];
    double *sm_aggregate = (double *)(sm + BATCH_SIZE_WORD);


    for(int batchId = 0; batchId < n_batch; batchId++){

        
        for(int i = 0; i < BATCH_SIZE; i += N_BODY_COPY_PER_PASS){

            int global_offset = batchId * BATCH_SIZE_WORD;
            int local_offset = i * BODY_SIZE_WORD + tid;
            int idx = global_offset + local_offset;

            if(idx < n * BODY_SIZE_WORD){
                sm[local_offset] = ((WORD *)bodyArray)[idx];
            }
        }

        __syncthreads();

        int bodyId_other = batchId * BATCH_SIZE + threadIdx.y;
        
        if ((bodyId_other != bodyId_this) && (bodyId_other < n)){
            
            double mj = ((Body *)sm)[threadIdx.y].m;
     
            if (((Body *)sm)[threadIdx.y].isDevice == 1) {
                mj = gravity_device_mass(mj, step * dt);
            }

            dx = ((Body *)sm)[threadIdx.y].qx - qx;
            dy = ((Body *)sm)[threadIdx.y].qy - qy;
            dz = ((Body *)sm)[threadIdx.y].qz - qz;

            // if(bodyId_this == 0) start_time = clock();

            double dist3 = pow(dx * dx + dy * dy + dz * dz + eps * eps, 1.5);

            ax += G * mj * dx / dist3;    
            ay += G * mj * dy / dist3;    
            az += G * mj * dz / dist3; 

            // if(bodyId_this == 0){
            //     stop_time = clock();
            //     runtime = (int)(stop_time - start_time);
            //     printf("dt: %d\n", runtime);
            // } 
        }
    }

    sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 0] = ax * dt;
    sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 1] = ay * dt;
    sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 2] = az * dt;

    __syncthreads();

   
                      
    for(int binSize = 2; binSize <= blockDim.y; binSize = binSize << 1){

        if((threadIdx.y & (binSize - 1)) == 0){

            sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 0] += \
                sm_aggregate[(threadIdx.y + (binSize >> 1)) * (3 * blockDim.x) \
                        + 3 * threadIdx.x + 0];
            
            sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 1] += \
                sm_aggregate[(threadIdx.y + (binSize >> 1)) * (3 * blockDim.x) \
                        + 3 * threadIdx.x + 1];
            
            sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 2] += \
                sm_aggregate[(threadIdx.y + (binSize >> 1)) * (3 * blockDim.x) \
                        + 3 * threadIdx.x + 2];

        }
        __syncthreads();
    }



    if((threadIdx.y < 3) && (bodyId_this < n)){
        
        double *v_ptr = (double *)&(bodyArray[bodyId_this].vx);
        double *q_ptr_update = (double *)&(bodyArray_update[bodyId_this].qx);
        double *v_ptr_update = (double *)&(bodyArray_update[bodyId_this].vx);
        double *q_ptr_update_sm = sm_aggregate + (3 * blockDim.x + 3 * threadIdx.x);
   
        q_ptr_update_sm[0] = qx;
        q_ptr_update_sm[1] = qy;
        q_ptr_update_sm[2] = qz;
        
        double vi = v_ptr[threadIdx.y];
        vi += sm_aggregate[3 * threadIdx.x + threadIdx.y];

        v_ptr_update[threadIdx.y] = vi;
        q_ptr_update[threadIdx.y] = q_ptr_update_sm[threadIdx.y] + vi * dt;
        
    }
}




__global__ void kernel_problem3(int step, int n_batch, int n, int asteroidId,
            Body *bodyArray, Body *bodyArray_update, BYTE *missile_cost, BYTE *success){


    int bodyId_this = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    double ax = 0, ay = 0, az = 0, dx, dy, dz;
    double qx, qy, qz;
   
    if(bodyId_this < n){
        qx = bodyArray[bodyId_this].qx;
        qy = bodyArray[bodyId_this].qy;
        qz = bodyArray[bodyId_this].qz;
    }

    if((bodyId_this == asteroidId) && (threadIdx.y == 0)){

        // check asteroid hit planet.
        dx = bodyArray[0].qx - qx;
        dy = bodyArray[0].qy - qy;
        dz = bodyArray[0].qz - qz;

        if (dx * dx + dy * dy + dz * dz < planet_radius * planet_radius) {
            *((int *)success) = 0;
        }
    }



    __shared__ WORD sm[BATCH_SIZE_WORD + N_THRD_PER_BLK_Y * 3 * 2 * N_THRD_PER_BLK_X];
    double *sm_aggregate = (double *)(sm + BATCH_SIZE_WORD);


    for(int batchId = 0; batchId < n_batch; batchId++){

        
        for(int i = 0; i < BATCH_SIZE; i += N_BODY_COPY_PER_PASS){

            int global_offset = batchId * BATCH_SIZE_WORD;
            int local_offset = i * BODY_SIZE_WORD + tid;
            int idx = global_offset + local_offset;

            if(idx < n * BODY_SIZE_WORD){
                sm[local_offset] = ((WORD *)bodyArray)[idx];
            }
        }

        __syncthreads();

        int bodyId_other = batchId * BATCH_SIZE + threadIdx.y;
        
        if ((bodyId_other != bodyId_this) && (bodyId_other < n)){
            
            double mj = ((Body *)sm)[threadIdx.y].m;
     
            if (((Body *)sm)[threadIdx.y].isDevice == 1) {
                mj = gravity_device_mass(mj, step * dt);
            }

            dx = ((Body *)sm)[threadIdx.y].qx - qx;
            dy = ((Body *)sm)[threadIdx.y].qy - qy;
            dz = ((Body *)sm)[threadIdx.y].qz - qz;

            double dist3 = pow(dx * dx + dy * dy + dz * dz + eps * eps, 1.5);

            ax += G * mj * dx / dist3;    
            ay += G * mj * dy / dist3;    
            az += G * mj * dz / dist3; 
        }
    }

    sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 0] = ax * dt;
    sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 1] = ay * dt;
    sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 2] = az * dt;

    __syncthreads();

   
                      
    for(int binSize = 2; binSize <= blockDim.y; binSize = binSize << 1){

        if((threadIdx.y & (binSize - 1)) == 0){

            sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 0] += \
                sm_aggregate[(threadIdx.y + (binSize >> 1)) * (3 * blockDim.x) \
                        + 3 * threadIdx.x + 0];
            
            sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 1] += \
                sm_aggregate[(threadIdx.y + (binSize >> 1)) * (3 * blockDim.x) \
                        + 3 * threadIdx.x + 1];
            
            sm_aggregate[threadIdx.y * (3 * blockDim.x) + 3 * threadIdx.x + 2] += \
                sm_aggregate[(threadIdx.y + (binSize >> 1)) * (3 * blockDim.x) \
                        + 3 * threadIdx.x + 2];

        }
        __syncthreads();
    }


    double *q_ptr_update_sm = sm_aggregate + (3 * blockDim.x + 3 * threadIdx.x);

    if((threadIdx.y < 3) && (bodyId_this < n)){
        
        double *v_ptr = (double *)&(bodyArray[bodyId_this].vx);
        double *q_ptr_update = (double *)&(bodyArray_update[bodyId_this].qx);
        double *v_ptr_update = (double *)&(bodyArray_update[bodyId_this].vx);
        
        q_ptr_update_sm[0] = qx;
        q_ptr_update_sm[1] = qy;
        q_ptr_update_sm[2] = qz;
        
        double vi = v_ptr[threadIdx.y];
        vi += sm_aggregate[3 * threadIdx.x + threadIdx.y];

        q_ptr_update_sm[threadIdx.y] += vi * dt;

        v_ptr_update[threadIdx.y] = vi;
        q_ptr_update[threadIdx.y] = q_ptr_update_sm[threadIdx.y];
        
    }

    __syncthreads();

    
    // check missile hit device.
    if((bodyId_this == 0) && (threadIdx.y == 0)){ 

        if(bodyArray[1].m != 0){
            
            dx = q_ptr_update_sm[0] - q_ptr_update_sm[3 + 0];
            dy = q_ptr_update_sm[1] - q_ptr_update_sm[3 + 1];
            dz = q_ptr_update_sm[2] - q_ptr_update_sm[3 + 2];

            double travel_dist = (step + 1) * dt * missile_speed;

            if (dx * dx + dy * dy + dz * dz < travel_dist * travel_dist){

                *((double *)missile_cost) = get_missile_cost((step + 1) * dt);

                bodyArray_update[1].m = 0;
            }
        }
        else if(bodyArray_update[1].m != 0){
            bodyArray_update[1].m = 0;
        }
      
    }


}




void problem1(hipStream_t stream, char* filename, double *min_dist_sq_ptr){

    Input input;
    Body *bodyArray1_dev, *bodyArray2_dev;
    BYTE *min_dist_sq_dev;

    read_input(filename, &input);

    for (int i = 0; i < input.n; i++) {
        if (input.bodyArray[i].isDevice == 1) input.bodyArray[i].m = 0;
    }

    // cudaSetDevice(0);

    hipMalloc(&bodyArray1_dev, input.n * sizeof(Body));
    hipMemcpy((BYTE *)bodyArray1_dev, (BYTE *)(input.bodyArray),
                            input.n * sizeof(Body), hipMemcpyHostToDevice);

    hipMalloc(&bodyArray2_dev, input.n * sizeof(Body));
    hipMemcpy((BYTE *)bodyArray2_dev, (BYTE *)(input.bodyArray),
                            input.n * sizeof(Body), hipMemcpyHostToDevice);

    double min_dist_sq_host = std::numeric_limits<double>::infinity();
    double min_dist_host;

    hipMalloc(&min_dist_sq_dev, sizeof(double));
    hipMemcpyAsync(min_dist_sq_dev, (BYTE *)&min_dist_sq_host,
                                    sizeof(double), hipMemcpyHostToDevice);

    int n_block = input.n / N_THRD_PER_BLK_X + 1;
    dim3 nThreadsPerBlock(N_THRD_PER_BLK_X, N_THRD_PER_BLK_Y, 1);

    int n_batch = input.n / BATCH_SIZE;
    if(n_batch * BATCH_SIZE < input.n) n_batch += 1;

    // auto start = high_resolution_clock::now();

    for (int step = 1; step <= n_steps + 1; step++) {

        kernel_problem1<<<n_block, nThreadsPerBlock, 0, stream>>>\
                (step, n_batch, input.n, input.planetId, input.asteroidId, 
                             bodyArray1_dev, bodyArray2_dev, min_dist_sq_dev);
        
        Body *tmp = bodyArray1_dev;
        bodyArray1_dev = bodyArray2_dev;
        bodyArray2_dev = tmp;
    }


    // auto stop = high_resolution_clock::now();
    // auto duration = duration_cast<microseconds>(stop - start);
    // cout<<"problem 1 time: "<<duration.count() / 1000000. <<" sec"<<endl;

    hipMemcpyAsync((BYTE *)min_dist_sq_ptr, min_dist_sq_dev, 
                                    sizeof(double), hipMemcpyDeviceToHost);    

}


void problem2(hipStream_t stream, char* filename, int *hit_time_step_ptr){

    Input input;
    Body *bodyArray1_dev, *bodyArray2_dev;
    BYTE *hit_time_step_dev;

    read_input(filename, &input);

    hipMalloc(&bodyArray1_dev, input.n * sizeof(Body));
    hipMemcpy((BYTE *)bodyArray1_dev, (BYTE *)(input.bodyArray),
                            input.n * sizeof(Body), hipMemcpyHostToDevice);

    hipMalloc(&bodyArray2_dev, input.n * sizeof(Body));
    hipMemcpy((BYTE *)bodyArray2_dev, (BYTE *)(input.bodyArray),
                            input.n * sizeof(Body), hipMemcpyHostToDevice);

    int  hit_time_step_host = -2;
 
    hipMalloc(&hit_time_step_dev, sizeof(int));
    hipMemcpyAsync(hit_time_step_dev, (BYTE *)&hit_time_step_host,
                                    sizeof(int), hipMemcpyHostToDevice);

    int n_block = input.n / N_THRD_PER_BLK_X + 1;
    dim3 nThreadsPerBlock(N_THRD_PER_BLK_X, N_THRD_PER_BLK_Y, 1);

    int n_batch = input.n / BATCH_SIZE;
    if(n_batch * BATCH_SIZE < input.n) n_batch += 1;




    auto start = high_resolution_clock::now();

    for (int step = 1; step <= n_steps + 1; step++) {

        kernel_problem2<<<n_block, nThreadsPerBlock, 0, stream>>>\
                (step, n_batch, input.n, input.planetId, input.asteroidId, 
                             bodyArray1_dev, bodyArray2_dev, hit_time_step_dev);

        Body *tmp = bodyArray1_dev;
        bodyArray1_dev = bodyArray2_dev;
        bodyArray2_dev = tmp;

        if((step & (16 - 1)) == 0){
            hipMemcpyAsync((BYTE *)hit_time_step_ptr, hit_time_step_dev, 
                                            sizeof(int), hipMemcpyDeviceToHost);
            // print();
            if(*hit_time_step_ptr != -2) break;
        }

    }


    hipDeviceSynchronize();

    hipMemcpy((BYTE *)hit_time_step_ptr, hit_time_step_dev, 
                                    sizeof(int), hipMemcpyDeviceToHost);

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout<<"problem 2 time: "<<duration.count() / 1000000. <<" sec"<<endl;

}



void swapBody(Input *input, int idx1, int idx2){
    if(idx1 == idx2) return;

    Body tmpBody = input->bodyArray[idx1];
    input->bodyArray[idx1] = input->bodyArray[idx2];
    input->bodyArray[idx2] = tmpBody;

    int tmpId = input->id_map[idx1];
    input->id_map[idx1] = input->id_map[idx2];
    input->id_map[idx2] = tmpId;
}


void problem3(hipStream_t stream, char* filename, int hit_time_step, 
                                        int *gravity_device_id_ptr, double *missile_cost_ptr){

    Input input;
    Body *bodyArray1_dev, *bodyArray2_dev;

    read_input(filename, &input);
    swapBody(&input, input.planetId, 0);

    hipMalloc(&bodyArray1_dev, input.n * sizeof(Body));
    // cudaMemcpy((BYTE *)bodyArray1_dev, (BYTE *)(input.bodyArray),
    //                         input.n * sizeof(Body), cudaMemcpyHostToDevice);

    hipMalloc(&bodyArray2_dev, input.n * sizeof(Body));
    // cudaMemcpy((BYTE *)bodyArray2_dev, (BYTE *)(input.bodyArray),
    //                         input.n * sizeof(Body), cudaMemcpyHostToDevice);

    BYTE *missile_cost_dev;
    double missile_cost_host;
    hipMalloc(&missile_cost_dev, sizeof(double));

    BYTE *success_dev;
    int success_host;
    hipMalloc(&success_dev, sizeof(int));



    int n_block = input.n / N_THRD_PER_BLK_X + 1;
    dim3 nThreadsPerBlock(N_THRD_PER_BLK_X, N_THRD_PER_BLK_Y, 1);

    int n_batch = input.n / BATCH_SIZE;
    if(n_batch * BATCH_SIZE < input.n) n_batch += 1;


    int gravity_device_id_min = -1;
    double missile_cost_min = std::numeric_limits<double>::infinity();


    if(hit_time_step != -2){

        for(int i = 0; i < input.n; i++){

            if(input.bodyArray[i].isDevice != 1) continue;
            if(input.bodyArray[i].m == 0) continue;

            int gravity_device_id = i;
            swapBody(&input, gravity_device_id, 1);


            success_host = 1;
            hipMemcpy(success_dev, (BYTE *)&success_host,
                                sizeof(int), hipMemcpyHostToDevice);            

            hipMemcpy((BYTE *)bodyArray1_dev, (BYTE *)(input.bodyArray),
                                    input.n * sizeof(Body), hipMemcpyHostToDevice);

            hipMemcpy((BYTE *)bodyArray2_dev, (BYTE *)(input.bodyArray),
                                    input.n * sizeof(Body), hipMemcpyHostToDevice);


            for (int step = 1; step <= n_steps + 1; step++) {

                kernel_problem3<<<n_block, nThreadsPerBlock, 0, stream>>>\
                        (step, n_batch, input.n, input.asteroidId, 
                        bodyArray1_dev, bodyArray2_dev, missile_cost_dev, success_dev);


                if((step & (16 - 1)) == 0){
                    hipMemcpyAsync((BYTE *)&success_host, success_dev, 
                                                    sizeof(int), hipMemcpyDeviceToHost);

                    if(success_host != 1) break;
                } 

                Body *tmp = bodyArray1_dev;
                bodyArray1_dev = bodyArray2_dev;
                bodyArray2_dev = tmp;
               
                
            }

            hipDeviceSynchronize();

            hipMemcpy((BYTE *)&success_host, success_dev, 
                                            sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy((BYTE *)&missile_cost_host, missile_cost_dev, 
                                            sizeof(double), hipMemcpyDeviceToHost);


            
            if(success_host == 1){
                if(missile_cost_host < missile_cost_min){                   
                    missile_cost_min = missile_cost_host;
                    gravity_device_id_min = gravity_device_id;
                }
            }

            // cudaMemcpy((BYTE *)bodyArray1_dev, (BYTE *)(input.bodyArray),
            //                         input.n * sizeof(Body), cudaMemcpyHostToDevice);

            // cudaMemcpy((BYTE *)bodyArray2_dev, (BYTE *)(input.bodyArray),
            //                         input.n * sizeof(Body), cudaMemcpyHostToDevice);

            swapBody(&input, gravity_device_id, 1);
        }

    }


                            
    if(gravity_device_id_min == -1){
        *gravity_device_id_ptr = -1;
        *missile_cost_ptr = 0;
    }
    else{  
        *gravity_device_id_ptr = gravity_device_id_min;
        *missile_cost_ptr = missile_cost_min;
    }


}





int hit_time_step;
double min_dist;
int gravity_device_id;
double missile_cost;

int main(int argc, char **argv)
{

    hipSetDevice(0);
    hipStream_t stream0[2];
    for (int i = 0; i < 2; ++i) hipStreamCreate(&stream0[i]);
    
    hipSetDevice(1);
    hipStream_t stream1[2];
    for (int i = 0; i < 2; ++i) hipStreamCreate(&stream1[i]);
    




    // cudaSetDevice(0);
    // double min_dist_sq;
    // problem1(stream0[0], argv[1], &min_dist_sq);
    // cudaDeviceSynchronize();
    // min_dist = sqrt(min_dist_sq);
    // printf("min_dist: %f\n", min_dist);




    // cudaSetDevice(0);
    // problem2(stream0[1], argv[1], &hit_time_step);
    // printf("hit_time_step: %d\n", hit_time_step);



    hit_time_step = 10;
    hipSetDevice(0);

    auto start = high_resolution_clock::now();

    problem3(stream0[1], argv[1], hit_time_step, &gravity_device_id, &missile_cost);

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout<<"problem 3 time: "<<duration.count() / 1000000. <<" sec"<<endl;

    printf("gravity_device_id: %d, missile_cost: %f\n", gravity_device_id, missile_cost);
  





    return 0;
}